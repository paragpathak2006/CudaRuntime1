#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <fstream>
#include <iostream>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>

#include "Geometry/Point.h"
#include "Containers/Space_map2.h"
#include "Input_output/Loader.h"
#include "Geometry/Point.h"
#include "Thrust_lib/thrust_dist.h"
#include "Thrust_lib/unsigned_distance_function.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ 
void addKernel(int *c, const int *a, const int *b){
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
#define _MINIMUM_(A,B) A < B ? A : B
#define _LINEAR_INDEX_(i,j,k,dim) i + j * dim + k * dim * dim
__global__
void calculate_min_dist(
    const Bucket* buckets, const Point_index* indexes, const Point* points, double* min_distances,
    Point target, double beta2, int bucket_count, int dim,
    int i0, int j0, int k0)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;

    if (i >= dim || j >= dim || k >= dim)
        return;

    int bucket_index = _HASH_(i0 + i, j0 + j, k0 + k) % bucket_count;

    int first = buckets[bucket_index].first;
    int count = buckets[bucket_index].count;
    double min_distance = beta2, dist;

    for (size_t iter = first; iter < count; iter++)
    {
        const Point_index& p = indexes[iter];
        if (p.x == i && p.y == j && p.z == k)
            dist = _DISTANCE_(points[p.index], target);

        min_distance = _MINIMUM_(min_distance, dist);
    }
    min_distances[_LINEAR_INDEX_(i, j, k, dim)] = min_distance;
}


#define _DIM3_(x) x,x,x
#define _CALC_BLOCK_DIM_(n,t) (n+t-1)/t
#define _MAP_INDEX_(x,y) round(x/y)
#define _CAST_(P) thrust::raw_pointer_cast(P.data())
#define _RAW_CAST_(P,Q,R,S) _CAST_(P) ,_CAST_(Q) , _CAST_(R) , _CAST_(S)
double custom_hash_map_implementation(const Points& points, const Point& target, double map_size,double beta)
{
    Space_map2 space_map(/* with input points as */ points, /* map_size as */ map_size);
    space_map.generate_cuda_hashmap();
    auto target_index = Point_index(target, map_size);
    auto beta2 = beta * beta;
    auto bucket_count = space_map.buckets.size();
    int max_index = _MAP_INDEX_(beta,map_size);
    max_index = max_index + max_index % 2;

    int num_threads = 2 * max_index;
    int threads_dim = 4;
    int blocks_dim = _CALC_BLOCK_DIM_(num_threads,threads_dim);
    dim3 threads_per_block(_DIM3_(threads_dim));
    dim3 blocks_per_grid(_DIM3_(blocks_dim));

    thrust::device_vector<Bucket> buckets(space_map.buckets);
    thrust::device_vector<Point_index> point_indexes(space_map.point_indexes);
    thrust::device_vector<Point> Dpoints(points);
    thrust::device_vector<double> min_distances(num_threads * num_threads * num_threads);

    calculate_min_dist << <blocks_per_grid, threads_per_block >> > (
        _RAW_CAST_(buckets, point_indexes, Dpoints, min_distances),
        target, beta2, bucket_count, 2 * max_index,
        target_index.x - max_index, target_index.y - max_index, target_index.z - max_index
        );

    return thrust::reduce(_ITER_(min_distances), beta2, min_dist());

}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\naddWithCuda failed!\n\n");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaDeviceReset failed!\n\n");
        return 1;
    }


    cout << "******************************************************************" << endl << endl;
    cout << "******************************************************************" << endl << endl;
    cout << "******************************************************************" << endl << endl;



    Point target = { 0,1,1.2 };
    double beta = 2;
    double map_size = 0.5;
    // Points points = {Point(0,0,0),Point(0,0,1),Point(0,1,1),Point(0,1,0)};
    Points points;

    objl::Mesh mesh;
    get_mesh("piston.obj", mesh);
    get_points(mesh, points);

    int nearest_point1;
    int nearest_point2;
    int nearest_point3;

    cout << "Beta : " << beta << endl;
    cout << "Map_size : " << map_size << endl << endl;
    cout << "Target point : "; target.print();
    cout << "Points : " << endl;

    //    for(const Point& p : points) p.print();

    cout << "------------------------------------------------------" << endl;
    cout << endl;
    cout << "Unsigned_distance_space_map Debug log" << endl;
    float dist3 = unsigned_distance_space_map_cuda(points, target, beta, map_size, nearest_point3);
    cout << endl << endl;

    cout << "------------------------------------------------------" << endl;
    cout << endl;
    cout << "Unsigned_distance_space_map Debug log" << endl;
    float dist2 = unsigned_distance_space_map2(points, target, beta, map_size, nearest_point2);
    cout << endl << endl;

    cout << "******************************************************************" << endl << endl;

    cout << "Unsigned_distance_brute_force output" << endl;
    float dist1 = unsigned_distance_brute_force(points, target, beta, nearest_point1);
    print_output(dist1, nearest_point1, target, points);

    cout << "------------------------------------------------------" << endl << endl;
    cout << "Unsigned_distance_space_map output..." << endl;
    print_output(dist2, nearest_point2, target, points);

    cout << "------------------------------------------------------" << endl << endl;
    cout << "Unsigned_distance_space_map cuda output..." << endl;
    cout << "Cuda Distance is : " << dist3;

    return 0;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaSetDevice failed!  \n\nDo you have a CUDA-capable GPU installed?\n\n");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMalloc failed!\n\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMalloc failed!\n\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMalloc failed!\n\n");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMalloc failed!\n\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMemcpy failed!\n\n");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\naddKernel launch failed: %s\n\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaDeviceSynchronize returned error code %d after launching addKernel!\n\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMemcpy failed!\n\n");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}



