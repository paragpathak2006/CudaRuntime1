#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>

#include <fstream>
#include <iostream>
#include <vector>
#include "Point.h"
#include "Finder.h"
#include "Space_map2.h"
#include "Loader.h"


struct dist_sqxy
{
    const double x, y;
    dist_sqxy(double _x, double _y) : x(_x), y(_y) {}
    __host__ __device__
        double operator()(const double& X, const double& Y) const {
        return (X - x) * (X - x) + (Y - y) * (Y - y);
    }
};

struct dist_sqz
{
    const double z;
    dist_sqz(double _z) : z(_z) {}
    __host__ __device__
        double operator()(const double& Z, const double& Y) const {
        return (Z - z) * (Z - z) + Y;
    }
};

struct mim_dist
{
    __host__ __device__
        double operator()(const double& Z, const double& Y) const {
        return (Z < Y) ? Z : Y;
    }
};

typedef thrust::host_vector<double> Hvec;
typedef thrust::device_vector<double> Dvec;
double min_dist_calculation(const Hvec& Px, const Hvec& Py, const Hvec& Pz, const Point& target, const double& beta2);

double unsigned_distance_space_map_cuda(const Points& points, const Point& target, double beta, double map_size, int& nearest_point) {

//    cout << test_functor(target.z)(1.1, 1.2);
    nearest_point = -1;
    Space_map2 space_map(/* with input points as */ points, /* map_size as */ map_size);

    vector<int> point_indexes;
    int n = point_indexes.size();
    double beta2 = beta * beta;

    space_map.lookup_region(target, beta, point_indexes);

    Hvec X(n), Y(n), Z(n);
    for (const int& i : point_indexes)
    {
        X.push_back(points[i].x);
        Y.push_back(points[i].y);
        Z.push_back(points[i].z);
    }

    return min_dist_calculation(X, Y, Z, target, beta2);
}

double min_dist_calculation(const Hvec& Px, const Hvec& Py, const Hvec& Pz, const Point& target, const double& beta2) {
    Dvec X = Px, Y = Py, Z = Pz;

    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), dist_sqxy(target.x,target.y));
    thrust::transform(Z.begin(), Z.end(), Y.begin(), Y.begin(), dist_sqz(target.z));

    return thrust::reduce(Y.begin(), Y.end(), beta2, mim_dist());
}

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
float unsigned_distance_brute_force(const Points& points, const Point& target, double beta, int& nearest_point) {
    nearest_point = -1;
    float min_dist = target.dist(points[0]);
    int i = 0;
    double beta2 = beta * beta;
    for (const Point& p : points)
    {
        float dist = target.dist(p);
        if (dist < min_dist)
        {
            min_dist = dist;
            nearest_point = i;
        }
        i++;
    }
    return (min_dist > beta2) ? beta2 : min_dist;
}

double unsigned_distance_space_map(const Points& points, const Point& target, double beta, double map_size, int& nearest_point) {

    nearest_point = -1;
    Space_map::initialize_space_map(/* with input points as */ points,/* map_size as */ map_size, /*  and beta as */ beta);
    double unsigned_dist = Space_map::search_space_map(points, target, nearest_point);
    Space_map::make_empty();

    return unsigned_dist;
}

double unsigned_distance_space_map2(const Points& points, const Point& target, double beta, double map_size, int& nearest_point) {

    nearest_point = -1;
    Space_map2 space_map(/* with input points as */ points, /* map_size as */ map_size);
    double unsigned_dist = space_map.search_space_map(points, target, beta, nearest_point);
    space_map.make_empty();
    return unsigned_dist;
}

void print_output(float dist, int nearest_point, const Point& target, const Points& points) {
    cout << "Unsigned distance : " << sqrt(dist) << endl;
    cout << "Target point : "; target.print();
    cout << "Nearest point : ";
    if (nearest_point >= 0) points[nearest_point].print();
    else cout << "Point not found!" << endl;

    cout << endl << endl;
}


int main()
{

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\naddWithCuda failed!\n\n");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaDeviceReset failed!\n\n");
        return 1;
    }


    cout << "******************************************************************" << endl << endl;
    cout << "******************************************************************" << endl << endl;
    cout << "******************************************************************" << endl << endl;



    Point target = { 0,1,1.2 };
    double beta = 2;
    double map_size = 0.5;
    // Points points = {Point(0,0,0),Point(0,0,1),Point(0,1,1),Point(0,1,0)};
    Points points;

    objl::Mesh mesh;
    get_mesh("piston.obj", mesh);
    get_points(mesh, points);

    int nearest_point1;
    int nearest_point2;
    int nearest_point3;

    cout << "Beta : " << beta << endl;
    cout << "Map_size : " << map_size << endl << endl;
    cout << "Target point : "; target.print();
    cout << "Points : " << endl;

    //    for(const Point& p : points) p.print();

    cout << "------------------------------------------------------" << endl;
    cout << endl;
    cout << "Unsigned_distance_space_map Debug log" << endl;
    float dist3 = unsigned_distance_space_map_cuda(points, target, beta, map_size, nearest_point3);
    cout << endl << endl;

    cout << "------------------------------------------------------" << endl;
    cout << endl;
    cout << "Unsigned_distance_space_map Debug log" << endl;
    float dist2 = unsigned_distance_space_map2(points, target, beta, map_size, nearest_point2);
    cout << endl << endl;

    cout << "******************************************************************" << endl << endl;

    cout << "Unsigned_distance_brute_force output" << endl;
    float dist1 = unsigned_distance_brute_force(points, target, beta, nearest_point1);
    print_output(dist1, nearest_point1, target, points);

    cout << "------------------------------------------------------" << endl << endl;
    cout << "Unsigned_distance_space_map output..." << endl;
    print_output(dist2, nearest_point2, target, points);

    cout << "------------------------------------------------------" << endl << endl;
    cout << "Unsigned_distance_space_map cuda output..." << endl;
    cout << "Cuda Distance is : " << dist3;

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaSetDevice failed!  \n\nDo you have a CUDA-capable GPU installed?\n\n");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMalloc failed!\n\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMalloc failed!\n\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMalloc failed!\n\n");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMalloc failed!\n\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMemcpy failed!\n\n");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\naddKernel launch failed: %s\n\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaDeviceSynchronize returned error code %d after launching addKernel!\n\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMemcpy failed!\n\n");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
