#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <fstream>
#include <iostream>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>


#include "Geometry/Point.h"
#include "Containers/Space_map2.h"
#include "Input_output/Loader.h"
#include "Geometry/Point.h"
#include "Thrust_lib/thrust_dist.h"
#include "Thrust_lib/unsigned_distance_function.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ 
void addKernel(int *c, const int *a, const int *b){
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
#define _CAST_(P) thrust::raw_pointer_cast(P.data())
#define _RAW_CAST_(P,Q,R,S) _CAST_(P) ,_CAST_(Q) , _CAST_(R) , _CAST_(S)  

__global__
void calculate_min_dist(
    Bucket* buckets, Point_index* indexes, Point* points, double* min_distances,
    Point target, double beta2, int bucket_count, int imax, 
    int i0, int j0, int k0);

double custom_hash_map_implementation(const Points& points, const Point& target, double map_size,double beta) 
{
    Space_map2 space_map(/* with input points as */ points, /* map_size as */ map_size);
    space_map.generate_cuda_hashmap();
    auto pi = Point_index(target, map_size);
    auto beta2 = beta * beta;
    auto bucket_count = space_map.buckets.size();
    int max_index = round(beta / map_size);
    max_index = max_index + max_index % 2;
    //int n = 8 * max_index * max_index * max_index;
    //int threads_per_block = 256;
    //int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;

    int n = 2 * max_index;
    int threads_per_block_x = 4;
    int blocks_per_grid_x = (n + threads_per_block_x - 1) / threads_per_block_x;

    dim3 threads_per_block(threads_per_block_x, threads_per_block_x, threads_per_block_x);
    dim3 blocks_per_grid(blocks_per_grid_x, blocks_per_grid_x, blocks_per_grid_x);

    thrust::device_vector<Bucket> buckets(space_map.buckets);
    thrust::device_vector<Point_index> point_indexes(space_map.point_indexes);
    thrust::device_vector<Point> Dpoints(points);
    thrust::device_vector<double> min_distances(n*n*n);

    calculate_min_dist << <blocks_per_grid, threads_per_block >> > (
        _RAW_CAST_(buckets, point_indexes, Dpoints, min_distances),
        target, beta2, bucket_count, 2 * max_index,
        pi.x - max_index, pi.y - max_index, pi.z - max_index
        );

    return thrust::reduce(_ITER_(min_distances), beta2, min_dist());

}

__device__
size_t hash_of_point_index(int i, int j, int k) {
    return (i * 18397) + (j * 20483) + (k * 29303);
}
#define _DIS_(P,Q) (P.x-Q.x)*(P.x-Q.x) + (P.y-Q.y)*(P.y-Q.y) + (P.z-Q.z)*(P.z-Q.z) 
__global__ 
void calculate_min_dist(
    Bucket* buckets, Point_index* indexes, Point* points, double* min_distances,
    Point target, double beta2, int bucket_count, int n,
    int i0,int j0,int k0)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;

    if (i >= n || j >= n || k >= n)
        return;

    //int temp = threadIdx.x / imax;
    //int i = threadIdx.x % imax + i0;
    //int j = temp % jmax + j0;
    //int k = temp / jmax + k0;

    int bucket_i = hash_of_point_index(i0 + i, j0 + j, k0 + k) % bucket_count;

    int first = buckets[bucket_i].first;
    int count = buckets[bucket_i].count;
     
    double min_distance, dist;
    min_distance = beta2;
    for (size_t i = first; i < count; i++)
    {
        if (indexes[i].x == i && indexes[i].y == j && indexes[i].z == k)
            dist = _DIS_(points[indexes[i].index], target);
            min_distance = (min_distance < dist) ? min_distance : dist;
    };

    min_distances[i + j*n + k*n*n] = min_distance;
}

int main()
{
    int n = 5;
    int p = 5 / 3;
    cout << 5.0 / 3.0<<endl;
    cout << p << endl;
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\naddWithCuda failed!\n\n");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaDeviceReset failed!\n\n");
        return 1;
    }


    cout << "******************************************************************" << endl << endl;
    cout << "******************************************************************" << endl << endl;
    cout << "******************************************************************" << endl << endl;



    Point target = { 0,1,1.2 };
    double beta = 2;
    double map_size = 0.5;
    // Points points = {Point(0,0,0),Point(0,0,1),Point(0,1,1),Point(0,1,0)};
    Points points;

    objl::Mesh mesh;
    get_mesh("piston.obj", mesh);
    get_points(mesh, points);

    int nearest_point1;
    int nearest_point2;
    int nearest_point3;

    cout << "Beta : " << beta << endl;
    cout << "Map_size : " << map_size << endl << endl;
    cout << "Target point : "; target.print();
    cout << "Points : " << endl;

    //    for(const Point& p : points) p.print();

    cout << "------------------------------------------------------" << endl;
    cout << endl;
    cout << "Unsigned_distance_space_map Debug log" << endl;
    float dist3 = unsigned_distance_space_map_cuda(points, target, beta, map_size, nearest_point3);
    cout << endl << endl;

    cout << "------------------------------------------------------" << endl;
    cout << endl;
    cout << "Unsigned_distance_space_map Debug log" << endl;
    float dist2 = unsigned_distance_space_map2(points, target, beta, map_size, nearest_point2);
    cout << endl << endl;

    cout << "******************************************************************" << endl << endl;

    cout << "Unsigned_distance_brute_force output" << endl;
    float dist1 = unsigned_distance_brute_force(points, target, beta, nearest_point1);
    print_output(dist1, nearest_point1, target, points);

    cout << "------------------------------------------------------" << endl << endl;
    cout << "Unsigned_distance_space_map output..." << endl;
    print_output(dist2, nearest_point2, target, points);

    cout << "------------------------------------------------------" << endl << endl;
    cout << "Unsigned_distance_space_map cuda output..." << endl;
    cout << "Cuda Distance is : " << dist3;

    return 0;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaSetDevice failed!  \n\nDo you have a CUDA-capable GPU installed?\n\n");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMalloc failed!\n\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMalloc failed!\n\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMalloc failed!\n\n");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMalloc failed!\n\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMemcpy failed!\n\n");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\naddKernel launch failed: %s\n\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaDeviceSynchronize returned error code %d after launching addKernel!\n\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n\ncudaMemcpy failed!\n\n");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}



